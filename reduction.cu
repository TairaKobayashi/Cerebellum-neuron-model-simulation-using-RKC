#include "hip/hip_runtime.h"
#ifndef _REDUCE_KERNEL_H_
#define _REDUCE_KERNEL_H_

#include <stdio.h>
#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

// Utility class used to avoid linker errors with extern
// unsized shared memory arrays with templated type
template<class T>
struct SharedMemory
{
    __device__ inline operator       T *()
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }

    __device__ inline operator const T *() const
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }
};

// specialize for double to avoid unaligned memory
// access compile errors
template<>
struct SharedMemory<double>
{
    __device__ inline operator       double *()
    {
        extern __shared__ double __smem_d[];
        return (double *)__smem_d;
    }

    __device__ inline operator const double *() const
    {
        extern __shared__ double __smem_d[];
        return (double *)__smem_d;
    }
};



bool isPow2(unsigned int x)
{
    return ((x&(x-1))==0);
}

unsigned int nextPow2(unsigned int x)
{
    --x;
    x |= x >> 1;
    x |= x >> 2;
    x |= x >> 4;
    x |= x >> 8;
    x |= x >> 16;
    return ++x;
}

#ifndef MIN
#define MIN(x,y) ((x < y) ? x : y)
#endif

/*
    This version adds multiple elements per thread sequentially.  This reduces the overall
    cost of the algorithm while keeping the work complexity O(n) and the step complexity O(log n).
    (Brent's Theorem optimization)

    Note, this kernel needs a minimum of 64*sizeof(T) bytes of shared memory.
    In other words if blockSize <= 32, allocate 64*sizeof(T) bytes.
    If blockSize > 32, allocate blockSize*sizeof(T) bytes.
*/
template <class T, unsigned int blockSize, bool nIsPow2>
__global__ void
reduce6(T *g_idata, T *g_odata, unsigned int n)
{
    // Handle to thread block group
    cg::thread_block cta = cg::this_thread_block();
    T *sdata = SharedMemory<T>();

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockSize*2 + threadIdx.x;
    unsigned int gridSize = blockSize*2*gridDim.x;

    T mySum = 0;

    // we reduce multiple elements per thread.  The number is determined by the
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < n)
    {
        mySum += g_idata[i];

        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (nIsPow2 || i + blockSize < n)
            mySum += g_idata[i+blockSize];

        i += gridSize;
    }

    // each thread puts its local sum into shared memory
    sdata[tid] = mySum;
    cg::sync(cta);


    // do reduction in shared mem
    if ((blockSize >= 512) && (tid < 256))
    {
        sdata[tid] = mySum = mySum + sdata[tid + 256];
    }

    cg::sync(cta);

    if ((blockSize >= 256) &&(tid < 128))
    {
            sdata[tid] = mySum = mySum + sdata[tid + 128];
    }

    cg::sync(cta);

    if ((blockSize >= 128) && (tid <  64))
    {
       sdata[tid] = mySum = mySum + sdata[tid +  64];
    }

    cg::sync(cta);

    cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

    if (cta.thread_rank() < 32)
    {
        // Fetch final intermediate sum from 2nd warp
        if (blockSize >=  64) mySum += sdata[tid + 32];
        // Reduce final warp using shuffle
        for (int offset = tile32.size()/2; offset > 0; offset /= 2) 
        {
             mySum += tile32.shfl_down(mySum, offset);
        }
    }

    // write result for this block to global mem
    if (cta.thread_rank() == 0) g_odata[blockIdx.x] = mySum;
}

template <class T>
void
reduce(int size, int threads, int blocks,
       T *d_idata, T *d_odata)
{
    dim3 dimBlock(threads, 1, 1);
    dim3 dimGrid(blocks, 1, 1);

    // when there is only one warp per block, we need to allocate two warps
    // worth of shared memory so that we don't index shared memory out of bounds
    int smemSize = (threads <= 32) ? 2 * threads * sizeof(T) : threads * sizeof(T);

    if (isPow2(size))
    {
        switch (threads)
        {
            case 512:
                reduce6<T, 512, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                break;

            case 256:
                reduce6<T, 256, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                break;

            case 128:
                reduce6<T, 128, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                break;

            case 64:
                reduce6<T,  64, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                break;

            case 32:
                reduce6<T,  32, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                break;

            case 16:
                reduce6<T,  16, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                break;

            case  8:
                reduce6<T,   8, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                break;

            case  4:
                reduce6<T,   4, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                break;

            case  2:
                reduce6<T,   2, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                break;

            case  1:
                reduce6<T,   1, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                break;
        }
    }
    else
    {
        switch (threads)
        {
            case 512:
                reduce6<T, 512, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                break;

            case 256:
                reduce6<T, 256, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                break;

            case 128:
                reduce6<T, 128, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                break;

            case 64:
                reduce6<T,  64, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                break;

            case 32:
                reduce6<T,  32, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                break;

            case 16:
                reduce6<T,  16, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                break;

            case  8:
                reduce6<T,   8, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                break;

            case  4:
                reduce6<T,   4, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                break;

            case  2:
                reduce6<T,   2, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                break;

            case  1:
                reduce6<T,   1, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                break;
        }
    }

}

void getNumBlocksAndThreads(int n, int maxBlocks, int maxThreads, int &blocks, int &threads)
{
    static int init = 0;
    static int device;
    static hipDeviceProp_t prop;
    //get device capability, to avoid block/grid size exceed the upper bound
    if(init == 0){
        hipGetDevice(&device);
        hipGetDeviceProperties(&prop, device);
        init = 1;
        fprintf(stderr, "getNumBlocksAndThreads\n");
    }

    threads = (n < maxThreads*2) ? nextPow2((n + 1)/ 2) : maxThreads;
    blocks = (n + (threads * 2 - 1)) / (threads * 2);

    if ((float)threads*blocks > (float)prop.maxGridSize[0] * prop.maxThreadsPerBlock)
    {
        fprintf( stderr, "n is too large, please choose a smaller number!\n");
        exit(1);
    }

    if (blocks > prop.maxGridSize[0])
    {
        fprintf(stderr, "Grid size <%d> exceeds the device capability <%d>, set block size as %d (original %d)\n",
               blocks, prop.maxGridSize[0], threads*2, threads);

        blocks /= 2;
        threads *= 2;
    }

    blocks = MIN(maxBlocks, blocks);
}




// Instantiate the reduction function for 3 types
template void
reduce<int>(int size,int threads, int blocks,
 int *d_idata, int *d_odata);

template void
reduce<float>(int size,int threads, int blocks,
 float *d_idata, float *d_odata);

template void
reduce<double>(int size,int threads, int blocks,
 double *d_idata, double *d_odata);



template <class T>
void ParallelReduction( int n,
                     T *d_idata,
                     T *d_odata,
                     T* result)
{
    //static T gpu_result = 0;
    static int maxThreads = 256;  // number of threads per block
    static int maxBlocks = 64;
    //static int count = 0; // debug
    static T *tmp;


    //hipDeviceSynchronize();

    // sum partial block sums on GPU
    int s=n;

    while (s > 1)
    {
        int threads = 0, blocks = 0;
        getNumBlocksAndThreads(s, maxBlocks, maxThreads, blocks, threads);
        reduce<T>(s, threads, blocks, d_idata, d_odata);
        //hipMemcpy(d_idata, d_odata, s*sizeof(T), hipMemcpyDeviceToDevice);
        
        tmp = d_odata;
        d_odata = d_idata;
        d_idata = tmp;

        s = (s + (threads*2-1)) / (threads*2);
    }

    //if(needReadBack){
        hipMemcpy( result, tmp, sizeof(T), hipMemcpyDeviceToHost);
        //fprintf(stderr, "memcpy:count %d\n",++count);
    //}
    return;
}


// Instantiate the reduction function for 3 types
template void
ParallelReduction<int>( int n,
                     int *d_idata,
                     int *d_odata,
                     int *result);

template void
ParallelReduction<float>( int n,
                     float *d_idata,
                     float *d_odata,
                     float *result);

template void
ParallelReduction<double>( int n,
                     double *d_idata,
                     double *d_odata,
                     double*result);


#endif // #ifndef _REDUCE_KERNEL_H_

