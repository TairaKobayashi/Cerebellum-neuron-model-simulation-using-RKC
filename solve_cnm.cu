#include "hip/hip_runtime.h"
#include "solve_cnm.cuh"

//#include ""
#include <hipblas.h>
#include <hipsparse.h>
//#include <math.h>

__global__ static 
void reset_vec ( neuron_solve_t *d_solve, const int nc )
{
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  if ( id < nc ) 
  {
    d_solve -> b [ id ] = 0.0;
    d_solve -> vec [ cn_gamma2 ] [ id ] = 0.0;
    d_solve -> vec [ cn_ommega2 ] [ id ] = 0.0;
  }
}

__host__
static void cg_cusparse_crs(const int ngc, const int nnz, 
  const double *d_val, const int *d_col, const int *d_row, double *d_x, double *d_b)
{
  static double *d_r, *d_p, *d_ap;
  static double *_buffer;
  static int size = 0;
  if ( size < ngc ) {
    if ( size == 0 ) {
        hipFree ( d_r );
        hipFree ( d_p );
        hipFree ( d_ap );
        hipFree ( _buffer );
    }
    hipMalloc ( ( double ** ) &d_r,  ngc * sizeof ( double ) );
    hipMalloc ( ( double ** ) &d_p,  ngc * sizeof ( double ) );
    hipMalloc ( ( double ** ) &d_ap, ngc * sizeof ( double ) );
    hipMalloc ( ( double ** ) &_buffer, ngc * sizeof ( double ) );
    size = ngc;
  }


  //hipMalloc ( ( double ** ) &d_r,  ngc * sizeof ( double ) );
  //hipMalloc ( ( double ** ) &d_p,  ngc * sizeof ( double ) );
  //hipMalloc ( ( double ** ) &d_ap, ngc * sizeof ( double ) );
    
  double bnorm, rnorm_k, rnorm_k1;
  double alpha, beta, pap;
  double epsilon = 1.0e-15;
  double cp1 = 1.0;
  double c0 = 0.0;
  double cm1 = -1.0;
  
  hipblasStatus_t stat1;
  hipblasHandle_t handle1;
  hipsparseStatus_t stat2;
  hipsparseHandle_t handle2;
  hipsparseMatDescr_t descrA;
  stat1 = hipblasCreate(&handle1);
  stat2 = hipsparseCreate(&handle2);
  hipsparseCreateMatDescr(&descrA);
  hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
 
  stat1 = hipblasDscal(handle1, ngc, &c0, d_x, 1);				// x = 0
  stat1 = hipblasDcopy(handle1, ngc, d_b, 1, d_r, 1);			// r = b
  stat1 = hipblasDcopy(handle1, ngc, d_r, 1, d_p, 1);			// p = r
  stat1 = hipblasDdot(handle1, ngc, d_b, 1, d_b, 1, &bnorm);	// ||b||
  /**/
  for (int k = 0; k < 100; k++) {
    //stat2 = hipsparseDcsrmv(handle2, HIPSPARSE_OPERATION_NON_TRANSPOSE,
    //ngc, ngc, nnz, &cp1, descrA, d_val, d_row, d_col, d_p, &c0, d_ap);	// Ap
    stat2 = cusparseCsrmvEx(handle2,CUSPARSE_ALG_MERGE_PATH, HIPSPARSE_OPERATION_NON_TRANSPOSE,
			ngc, ngc, nnz, &cp1, HIP_R_64F, descrA, d_val, HIP_R_64F, d_row, d_col, d_p, HIP_R_64F, 
			&c0, HIP_R_64F, d_ap, HIP_R_64F, HIP_R_64F, _buffer );	// Ap
    stat1 = hipblasDdot(handle1, ngc, d_r, 1, d_r, 1, &rnorm_k);				// ||r_k||^2
    stat1 = hipblasDdot(handle1, ngc, d_p, 1, d_ap, 1, &pap);				// pAp		
    alpha = rnorm_k / pap;													// alpha		
    stat1 = hipblasDaxpy(handle1, ngc, &alpha, d_p, 1, d_x, 1);				// x += alpha * p
    alpha = -1.0 * alpha;
    stat1 = hipblasDaxpy(handle1, ngc, &alpha, d_ap, 1, d_r, 1);				// r -= alpha * ap
    stat1 = hipblasDdot(handle1, ngc, d_r, 1, d_r, 1, &rnorm_k1);			// ||r_k+1||^2
  
    if (sqrt(rnorm_k1) <= epsilon * sqrt(bnorm)) { break; }
  
    // p = r + beta * p
    beta = rnorm_k1 / rnorm_k;
    stat1 = hipblasDscal(handle1, ngc, &beta, d_p, 1);
    stat1 = hipblasDaxpy(handle1, ngc, &cp1, d_r, 1, d_p, 1);
  }
  hipsparseDestroyMatDescr(descrA);
  hipblasDestroy(handle1);
  hipsparseDestroy(handle2);//
      
  //hipFree ( d_r );
  //hipFree ( d_p );
  //hipFree ( d_ap );
}


//////////////////////////////// GR /////////////////////////////////
__global__ static
void add_mfgr_val ( neuron_solve_t *d_gr_solve, int *mfgr_comp, double *mfgr_elem, const int num_mfgr )
{
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  if ( id < num_mfgr )
  {
    int post_num = mfgr_comp [ post_comp * num_mfgr + id ];
	  double l_val = mfgr_elem [ mfgr_val * num_mfgr + id ];
    atomicAdd ( & ( d_gr_solve -> vec [ cn_gamma2  ] [ post_num ] ), 0.5 * l_val ); 
    atomicAdd ( & ( d_gr_solve -> vec [ cn_ommega2 ] [ post_num ] ), 0.5 * l_val * E_MFGR );
  }
}

__global__ static
void add_gogr_val ( neuron_t *d_gr, neuron_solve_t *d_gr_solve, 
                    int *gogr_comp, double *gogr_elem, const int num_gogr )
{
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  if ( id < num_gogr )
  {
    int post_num = gogr_comp [ post_comp * num_gogr + id ];
	  double l_val = gogr_elem [ gogr_val * num_gogr + id ];
    atomicAdd ( & ( d_gr_solve -> vec [ cn_gamma2  ] [ post_num ] ), 0.5 * l_val ); // no need atomicAdd ?
    atomicAdd ( & ( d_gr_solve -> vec [ cn_ommega2 ] [ post_num ] ), 0.5 * l_val * E_GOGR );
  }
}

__global__ 
void gr_cnm_vec_initialize ( neuron_t *d_gr, neuron_solve_t *d_gr_solve )
{
  double **elem = d_gr -> elem;
  double **cond = d_gr -> cond;
  double **ion  = d_gr -> ion;
  double **vec  = d_gr_solve -> vec;  
  //double *val         = d_gr_solve -> val;
  //double *val_ori     = d_gr_solve -> val_ori;
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  if ( id < d_gr -> nc) 
  {
    vec [ cn_gamma1 ] [ id ] = 
                  (   cond [ g_leak1 ] [ id ] + cond [ g_leak2 ] [ id ] + cond [ g_leak3 ] [ id ]
                    + cond [ g_Na ]  [ id ] * ion [ o_Na ] [ id ]
                    + cond [ g_Ca ]  [ id ] * ion [ ch_Ca ] [ id ] * ion [ ch_Ca ] [ id ] * ion [ ci_Ca ] [ id ]
                    + cond [ g_KV ]  [ id ] * ion [ n_KV ] [ id ] * ion [ n_KV ] [ id ] * ion [ n_KV ] [ id ] * ion [ n_KV ] [ id ]
                    + cond [ g_KIR ] [ id ] *  ion [ ir_KIR ] [ id ]
                    + cond [ g_KA ]  [ id ] *  ion [ a_KA ] [ id ] * ion [ a_KA ] [ id ] * ion [ a_KA ] [ id ] * ion [ b_KA ] [ id ]
                    + cond [ g_KCa ] [ id ] *  ion [ c_KCa ] [ id ]
                    + cond [ g_KM ]  [ id ] *  ion [ s_KM ] [ id ] ) / 2.0;
    vec [ cn_gamma2 ] [ id ] = 0.0;
    vec [ cn_ommega1 ] [ id ] = 
                  (   cond [ g_leak1 ] [ id ] * V_LEAK1_GR + cond [ g_leak2 ] [ id ] * V_LEAK2_GR + cond [ g_leak3 ] [ id ] * V_LEAK3_GR + elem [ i_ext ] [ id ]
                    + cond [ g_Na    ] [ id ] * V_Na_GR * ion [ o_Na   ] [ id ]
                    + cond [ g_Ca    ] [ id ] * V_Ca_GR * ion [ ch_Ca  ] [ id ] * ion [ ch_Ca ] [ id ] * ion [ ci_Ca ] [ id ]
                    + cond [ g_KV    ] [ id ] * V_K_GR  * ion [ n_KV   ] [ id ] * ion [ n_KV  ] [ id ] * ion [ n_KV  ] [ id ] * ion [ n_KV ] [ id ]
                    + cond [ g_KIR   ] [ id ] * V_K_GR  * ion [ ir_KIR ] [ id ]
                    + cond [ g_KA    ] [ id ] * V_K_GR  * ion [ a_KA   ] [ id ] * ion [ a_KA  ] [ id ] * ion [ a_KA  ] [ id ] * ion [ b_KA ] [ id ]
                    + cond [ g_KCa   ] [ id ] * V_K_GR  * ion [ c_KCa  ] [ id ]
                    + cond [ g_KM    ] [ id ] * V_K_GR  * ion [ s_KM   ] [ id ]  ) / 2.0;
    vec [ cn_ommega2 ] [ id ] = 0.0;
    vec [ cn_v_old ] [ id ] = elem [ v ] [ id ];
  }
  //for ( int i = 0; i < gr_solve -> nnz; i++ )  { val [ id ] /= 2.0; val_ori [ id ] = val [ id ]; } // to gr_solve.cu
}
__global__
static void gr_update_matrix ( neuron_t *d_gr, neuron_solve_t *d_gr_solve )
{
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  double **elem = d_gr -> elem;
  double **cond = d_gr -> cond;
  double **ion  = d_gr -> ion;
  double **vec    = d_gr_solve -> vec;
  double *val     = d_gr_solve -> val;
  double *val_ori = d_gr_solve -> val_ori;
  double *b       = d_gr_solve -> b;
  int    *col  = d_gr_solve -> col;
  int    *row  = d_gr_solve -> row;
  double DT = d_gr -> DT;
  if ( id < d_gr -> nc) 
  {
    vec [ cn_gamma2 ] [ id ] += 
    (   cond [ g_leak1 ] [ id ] + cond [ g_leak2 ] [ id ] + cond [ g_leak3 ] [ id ] 
      + cond [ g_Na ]  [ id ] * ion [ o_Na ] [ id ]
      + cond [ g_Ca ]  [ id ] * ion [ ch_Ca ] [ id ] * ion [ ch_Ca ] [ id ] * ion [ ci_Ca ] [ id ]
      + cond [ g_KV ]  [ id ] * ion [ n_KV ] [ id ] * ion [ n_KV ] [ id ] * ion [ n_KV ] [ id ] * ion [ n_KV ] [ id ]
      + cond [ g_KIR ] [ id ] *  ion [ ir_KIR ] [ id ]
      + cond [ g_KA ]  [ id ] *  ion [ a_KA ] [ id ] * ion [ a_KA ] [ id ] * ion [ a_KA ] [ id ] * ion [ b_KA ] [ id ]
      + cond [ g_KCa ] [ id ] *  ion [ c_KCa ] [ id ]
      + cond [ g_KM ]  [ id ] *  ion [ s_KM ] [ id ] ) / 2.0;
    vec [ cn_ommega2 ] [ id ] += 
    (   cond [ g_leak1 ] [ id ] * V_LEAK1_GR + cond [ g_leak2 ] [ id ] * V_LEAK2_GR + cond [ g_leak3 ] [ id ] * V_LEAK3_GR + elem [ i_ext ] [ id ]
      + cond [ g_Na    ] [ id ] * V_Na_GR * ion [ o_Na   ] [ id ]
      + cond [ g_Ca    ] [ id ] * V_Ca_GR * ion [ ch_Ca  ] [ id ] * ion [ ch_Ca ] [ id ] * ion [ ci_Ca ] [ id ]
      + cond [ g_KV    ] [ id ] * V_K_GR  * ion [ n_KV   ] [ id ] * ion [ n_KV  ] [ id ] * ion [ n_KV  ] [ id ] * ion [ n_KV ] [ id ]
      + cond [ g_KIR   ] [ id ] * V_K_GR  * ion [ ir_KIR ] [ id ]
      + cond [ g_KA    ] [ id ] * V_K_GR  * ion [ a_KA   ] [ id ] * ion [ a_KA  ] [ id ] * ion [ a_KA  ] [ id ] * ion [ b_KA ] [ id ]
      + cond [ g_KCa   ] [ id ] * V_K_GR  * ion [ c_KCa  ] [ id ]
      + cond [ g_KM    ] [ id ] * V_K_GR  * ion [ s_KM   ] [ id ]  ) / 2.0;

      int d = d_gr_solve -> dig [ id ];
      val [ d ] += ( elem [ Cm ] [ id ] / DT) + vec [ cn_gamma2 ] [ id ];
      b [ id ] = 0.0;
      for (int j = row [ id ]; j < row [ id + 1 ]; j++) {
          b [ id ] -= elem [ v ] [ col [ j ] ] * val_ori [ j ];
      }
      b [ id ] += (elem [ Cm ] [ id ] / DT - vec [ cn_gamma1 ] [ id ]) * elem [ v ][ id ] + vec [ cn_ommega1 ] [ id ] + vec [ cn_ommega2 ] [ id ];
      vec [ cn_ommega1 ] [ id ] = vec [ cn_ommega2 ] [ id ];
      vec [ cn_gamma1  ] [ id ] = vec [ cn_gamma2  ] [ id ];
  }
} 
__host__
void gr_solve_by_cnm ( neuron_t *d_gr, neuron_solve_t *d_gr_solve, 
                       neuron_t *p_gr, neuron_solve_t *p_gr_solve,
                       synapse_t *d_mfgr, synapse_t *d_gogr )
{  
  // global
  double **ion  = p_gr -> ion;
  double **elem = p_gr -> elem;
  int nc = p_gr -> nc;
  static int numThreadsPerBlock = p_gr_solve -> numThreadsPerBlock;
  static int numBlocks = p_gr_solve -> numBlocks;

  // update ion
  gr_Na_update_2order <<< numBlocks, numThreadsPerBlock >>>
   ( nc, elem [ v ], p_gr_solve -> vec [ cn_v_old ], CN_DT, elem [ compart ],
     ion [ o_Na ],  ion [ c1_Na ], ion [ c2_Na ], ion [ c3_Na ], ion [ c4_Na ], ion [ c5_Na ],
     ion [ i1_Na ], ion [ i2_Na ], ion [ i3_Na ], ion [ i4_Na ], ion [ i5_Na ], ion [ i6_Na ] );
  gr_update_ion_exp_imp <<< numBlocks, numThreadsPerBlock >>> ( d_gr, d_gr_solve, CN_DT );
  
  // reset val and b
  hipMemcpy ( p_gr_solve -> val,  p_gr_solve -> val_ori, p_gr_solve -> nnz * sizeof ( double ), hipMemcpyDeviceToDevice );
  reset_vec <<< numBlocks, numThreadsPerBlock >>> ( d_gr_solve, nc );

  // update val, b and v
  add_mfgr_val <<< ( d_mfgr -> n + 127 ) / numThreadsPerBlock, numThreadsPerBlock >>>
    ( d_gr_solve, d_mfgr -> comp, d_mfgr -> elem, d_mfgr -> n );
  add_gogr_val <<< ( d_gogr -> n + 127 ) / 128, 128 >>> 
    ( d_gr, d_gr_solve, d_gogr -> comp, d_gogr -> elem, d_gogr -> n );          //hipDeviceSynchronize();
  gr_update_matrix <<< numBlocks, numThreadsPerBlock >>> ( d_gr, d_gr_solve);  
  cg_cusparse_crs ( nc, p_gr_solve -> nnz, p_gr_solve -> val, p_gr_solve -> col, p_gr_solve -> row, p_gr -> elem [ v ], p_gr_solve -> b );
  
}

//////////////////////////////// GO /////////////////////////////////

__global__ static
void add_grgo_val ( neuron_t *d_go, neuron_solve_t *d_go_solve, 
                    int *grgo_comp, double *grgo_elem, const int num_grgo )
{
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  if ( id < num_grgo )
  {
    int post_num = grgo_comp [ post_comp * num_grgo + id ];
	  double l_val = grgo_elem [ grgo_val * num_grgo + id ];
    atomicAdd ( & ( d_go_solve -> vec [ cn_gamma2  ] [ post_num ] ), 0.5 * l_val ); // no need atomicAdd ?
    atomicAdd ( & ( d_go_solve -> vec [ cn_ommega2 ] [ post_num ] ), 0.5 * l_val * E_GRGO );
  }
}

__global__ 
void go_cnm_vec_initialize ( neuron_t *d_go, neuron_solve_t *d_go_solve )
{
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  if ( id < d_go -> nc) 
  {
    double **elem = d_go -> elem;
    double **cond = d_go -> cond;
    double **ion  = d_go -> ion;
    double **vec  = d_go_solve -> vec;  
    vec [ cn_gamma1 ] [ id ] = 
                     ( cond [ g_leak_go ] [ id ]
                     + cond [ g_NaT_go ] [ id ] * ion [ m_NaT_go ] [ id ] * ion [ m_NaT_go ] [ id ] * ion [ m_NaT_go ] [ id ] * ion [ h_NaT_go ] [ id ]
                     + cond [ g_NaR_go ] [ id ] * ion [ r_NaR_go ] [ id ] * ion [ s_NaR_go ] [ id ]
                     + cond [ g_NaP_go ] [ id ] * ion [ p_NaP_go ] [ id ]
                     + cond [ g_CaHVA_go ] [ id ] * ion [ ch_CaHVA_go ] [ id ] * ion [ ch_CaHVA_go ] [ id ] * ion [ ci_CaHVA_go ] [ id ]
                     + cond [ g_CaLVA_go ] [ id ] * ion [ cl_CaLVA_go ] [ id ] * ion [ cl_CaLVA_go ] [ id ] * ion [ cm_CaLVA_go ] [ id ]
                     + cond [ g_KAHP_go ] [ id ] * ( ion [ o1_KAHP_go ] [ id ] + ion [ o2_KAHP_go ] [ id ] )
                     + cond [ g_KV_go ] [ id ] * ion [ n_KV_go ] [ id ] * ion [ n_KV_go ] [ id ] * ion [ n_KV_go ] [ id ] * ion [ n_KV_go ] [ id ]
                     + cond [ g_KA_go ] [ id ] * ion [ a_KA_go ] [ id ] * ion [ a_KA_go ] [ id ] * ion [ a_KA_go ] [ id ] * ion [ b_KA_go ] [ id ]
                     + cond [ g_KC_go ] [ id ] * ion [ c_KC_go ] [ id ]
                     + cond [ g_Kslow_go ] [ id ] * ion [ sl_Kslow_go ] [ id ]
                     + cond [ g_HCN1_go ] [ id ] * ( ion [ hf_HCN1_go ] [ id ] + ion [ hs_HCN1_go ] [ id ] )
                     + cond [ g_HCN2_go ] [ id ] * ( ion [ hf_HCN2_go ] [ id ] + ion [ hs_HCN2_go ] [ id ] )
                     ) / 2.0;
    vec [ cn_gamma2 ] [ id ] = 0.0;
    vec [ cn_ommega1 ] [ id ] = 
                     ( cond [ g_leak_go ] [ id ] * V_LEAK_GO + elem [ i_ext ] [ id ]
                     + cond [ g_NaT_go ] [ id ] * V_Na_GO * ion [ m_NaT_go ] [ id ] * ion [ m_NaT_go ] [ id ] * ion [ m_NaT_go ] [ id ] * ion [ h_NaT_go ] [ id ]
                     + cond [ g_NaR_go ] [ id ] * V_Na_GO * ion [ r_NaR_go ] [ id ] * ion [ s_NaR_go ] [ id ]
                     + cond [ g_NaP_go ] [ id ] * V_Na_GO * ion [ p_NaP_go ] [ id ]
                     + cond [ g_CaHVA_go ] [ id ] * V_Ca_GO * ion [ ch_CaHVA_go ] [ id ] * ion [ ch_CaHVA_go ] [ id ] * ion [ ci_CaHVA_go ] [ id ]
                     + cond [ g_CaLVA_go ] [ id ] * ( d_go -> rev_ca2 [ id ] ) * ion [ cl_CaLVA_go ] [ id ] * ion [ cl_CaLVA_go ] [ id ] * ion [ cm_CaLVA_go ] [ id ]
                     + cond [ g_KAHP_go ] [ id ] * V_K_GO * ( ion [ o1_KAHP_go ] [ id ] + ion [ o2_KAHP_go ] [ id ] )
                     + cond [ g_KV_go ] [ id ] * V_K_GO  * ion [ n_KV_go ] [ id ] * ion [ n_KV_go ] [ id ] * ion [ n_KV_go ] [ id ] * ion [ n_KV_go ] [ id ]
                     + cond [ g_KA_go ] [ id ] * V_K_GO  * ion [ a_KA_go ] [ id ] * ion [ a_KA_go ] [ id ] * ion [ a_KA_go ] [ id ] * ion [ b_KA_go ] [ id ]
                     + cond [ g_KC_go ] [ id ] * V_K_GO * ion [ c_KC_go ] [ id ]
                     + cond [ g_Kslow_go ] [ id ] * V_K_GO * ion [ sl_Kslow_go ] [ id ]
                     + cond [ g_HCN1_go ] [ id ] * V_H_GO * ( ion [ hf_HCN1_go ] [ id ] + ion [ hs_HCN1_go ] [ id ] )
                     + cond [ g_HCN2_go ] [ id ] * V_H_GO * ( ion [ hf_HCN2_go ] [ id ] + ion [ hs_HCN2_go ] [ id ] ) 
                    ) / 2.0;
    vec [ cn_ommega2 ] [ id ] = 0.0;
    vec [ cn_v_old ] [ id ] = elem [ v ] [ id ];
  }
  //for ( int i = 0; i < go_solve -> nnz; i++ )  { val [ id ] /= 2.0; val_ori [ id ] = val [ id ]; } // to go_solve.cu
}
__global__
static void go_update_matrix ( neuron_t *d_go, neuron_solve_t *d_go_solve )
{
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  if ( id < d_go -> nc) 
  {
    double **elem = d_go -> elem;
    double **cond = d_go -> cond;
    double **ion  = d_go -> ion;
    double **vec    = d_go_solve -> vec;
    double *val     = d_go_solve -> val;
    double *val_ori = d_go_solve -> val_ori;
    double *b       = d_go_solve -> b;
    int    *col  = d_go_solve -> col;
    int    *row  = d_go_solve -> row;
    double DT = d_go -> DT;
    vec [ cn_gamma2 ] [ id ] += 
      ( cond [ g_leak_go ] [ id ] 
      + cond [ g_NaT_go ] [ id ] * ion [ m_NaT_go ] [ id ] * ion [ m_NaT_go ] [ id ] * ion [ m_NaT_go ] [ id ] * ion [ h_NaT_go ] [ id ]
      + cond [ g_NaR_go ] [ id ] * ion [ r_NaR_go ] [ id ] * ion [ s_NaR_go ] [ id ]
      + cond [ g_NaP_go ] [ id ] * ion [ p_NaP_go ] [ id ]
      + cond [ g_CaHVA_go ] [ id ] * ion [ ch_CaHVA_go ] [ id ] * ion [ ch_CaHVA_go ] [ id ] * ion [ ci_CaHVA_go ] [ id ]
      + cond [ g_CaLVA_go ] [ id ] * ion [ cl_CaLVA_go ] [ id ] * ion [ cl_CaLVA_go ] [ id ] * ion [ cm_CaLVA_go ] [ id ]
      + cond [ g_KAHP_go ] [ id ] * ( ion [ o1_KAHP_go ] [ id ] + ion [ o2_KAHP_go ] [ id ] )
      + cond [ g_KV_go ] [ id ] * ion [ n_KV_go ] [ id ] * ion [ n_KV_go ] [ id ] * ion [ n_KV_go ] [ id ] * ion [ n_KV_go ] [ id ]
      + cond [ g_KA_go ] [ id ] * ion [ a_KA_go ] [ id ] * ion [ a_KA_go ] [ id ] * ion [ a_KA_go ] [ id ] * ion [ b_KA_go ] [ id ]
      + cond [ g_KC_go ] [ id ] * ion [ c_KC_go ] [ id ]
      + cond [ g_Kslow_go ] [ id ] * ion [ sl_Kslow_go ] [ id ]
      + cond [ g_HCN1_go ] [ id ] * ( ion [ hf_HCN1_go ] [ id ] + ion [ hs_HCN1_go ] [ id ] )
      + cond [ g_HCN2_go ] [ id ] * ( ion [ hf_HCN2_go ] [ id ] + ion [ hs_HCN2_go ] [ id ] ) 
    ) / 2.0;
    vec [ cn_ommega2 ] [ id ] += 
      ( cond [ g_leak_go ] [ id ] * V_LEAK_GO + elem [ i_ext ] [ id ]
      + cond [ g_NaT_go ] [ id ] * V_Na_GO * ion [ m_NaT_go ] [ id ] * ion [ m_NaT_go ] [ id ] * ion [ m_NaT_go ] [ id ] * ion [ h_NaT_go ] [ id ]
      + cond [ g_NaR_go ] [ id ] * V_Na_GO * ion [ r_NaR_go ] [ id ] * ion [ s_NaR_go ] [ id ]
      + cond [ g_NaP_go ] [ id ] * V_Na_GO * ion [ p_NaP_go ] [ id ]
      + cond [ g_CaHVA_go ] [ id ] * V_Ca_GO * ion [ ch_CaHVA_go ] [ id ] * ion [ ch_CaHVA_go ] [ id ] * ion [ ci_CaHVA_go ] [ id ]
      + cond [ g_CaLVA_go ] [ id ] * ( d_go -> rev_ca2 [ id ] ) * ion [ cl_CaLVA_go ] [ id ] * ion [ cl_CaLVA_go ] [ id ] * ion [ cm_CaLVA_go ] [ id ]
      + cond [ g_KAHP_go ] [ id ] * V_K_GO * ( ion [ o1_KAHP_go ] [ id ] + ion [ o2_KAHP_go ] [ id ] )
      + cond [ g_KV_go ] [ id ] * V_K_GO  * ion [ n_KV_go ] [ id ] * ion [ n_KV_go ] [ id ] * ion [ n_KV_go ] [ id ] * ion [ n_KV_go ] [ id ]
      + cond [ g_KA_go ] [ id ] * V_K_GO  * ion [ a_KA_go ] [ id ] * ion [ a_KA_go ] [ id ] * ion [ a_KA_go ] [ id ] * ion [ b_KA_go ] [ id ]
      + cond [ g_KC_go ] [ id ] * V_K_GO * ion [ c_KC_go ] [ id ]
      + cond [ g_Kslow_go ] [ id ] * V_K_GO * ion [ sl_Kslow_go ] [ id ]
      + cond [ g_HCN1_go ] [ id ] * V_H_GO * ( ion [ hf_HCN1_go ] [ id ] + ion [ hs_HCN1_go ] [ id ] )
      + cond [ g_HCN2_go ] [ id ] * V_H_GO * ( ion [ hf_HCN2_go ] [ id ] + ion [ hs_HCN2_go ] [ id ] ) 
    ) / 2.0;

      int d = d_go_solve -> dig [ id ];
      val [ d ] += ( elem [ Cm ] [ id ] / DT) + vec [ cn_gamma2 ] [ id ];
      b [ id ] = 0.0;
      for (int j = row [ id ]; j < row [ id + 1 ]; j++) {
          b [ id ] -= elem [ v ] [ col [ j ] ] * val_ori [ j ];
      }
      b [ id ] += (elem [ Cm ] [ id ] / DT - vec [ cn_gamma1 ] [ id ]) * elem [ v ][ id ]
                 + vec [ cn_ommega1 ] [ id ] + vec [ cn_ommega2 ] [ id ];
      vec [ cn_ommega1 ] [ id ] = vec [ cn_ommega2 ] [ id ];
      vec [ cn_gamma1  ] [ id ] = vec [ cn_gamma2  ] [ id ];
  }
}
__host__
void go_solve_by_cnm ( neuron_t *d_go, neuron_solve_t *d_go_solve, 
                       neuron_t *p_go, neuron_solve_t *p_go_solve, synapse_t *d_grgo )
{  
  // global
  double **ion  = p_go -> ion;
  int nc = p_go -> nc;
  static int numThreadsPerBlock = p_go_solve -> numThreadsPerBlock;
  static int numBlocks = p_go_solve -> numBlocks;
  
  // update ion
  go_update_ion_exp_imp <<< numBlocks, numThreadsPerBlock >>> ( d_go, d_go_solve, CN_DT );
  go_KAHP_update_2order <<< numBlocks, numThreadsPerBlock >>> 
   ( p_go -> n, p_go -> elem [ Ca ], p_go -> ca_old, ion [ o1_KAHP_go ], ion [ o2_KAHP_go ], 
    ion [ c1_KAHP_go ], ion [ c2_KAHP_go ], ion [ c3_KAHP_go ], ion [ c4_KAHP_go ], CN_DT );  

  // reset val and b
  hipMemcpy ( p_go_solve -> val,  p_go_solve -> val_ori, p_go_solve -> nnz * sizeof ( double ), hipMemcpyDeviceToDevice );
  reset_vec <<< numBlocks, numThreadsPerBlock >>> ( d_go_solve, nc );
  
  // update val, b and v
  add_grgo_val <<< ( d_grgo -> n + 127 ) / 128, 128 >>> 
    ( d_go, d_go_solve, d_grgo -> comp, d_grgo -> elem, d_grgo -> n );          //hipDeviceSynchronize();
  go_update_matrix <<< numBlocks, numThreadsPerBlock >>> ( d_go, d_go_solve);
  cg_cusparse_crs( p_go -> nc, p_go_solve -> nnz, p_go_solve -> val, p_go_solve -> col, p_go_solve -> row, p_go -> elem [ v ], p_go_solve -> b );

}

//////////////////////////////// PKJ /////////////////////////////////

__global__ static
void add_mlipkj_val ( neuron_solve_t *d_pkj_solve, int *mlipkj_comp, double *mlipkj_elem, const int num_mlipkj )
{
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  if ( id < num_mlipkj )
  {
    int post_num = mlipkj_comp [ post_comp * num_mlipkj + id ];
	  double l_val = mlipkj_elem [ mlipkj_val * num_mlipkj + id ];
    atomicAdd ( & ( d_pkj_solve -> vec [ cn_gamma2  ] [ post_num ] ), 0.5 * l_val ); 
    atomicAdd ( & ( d_pkj_solve -> vec [ cn_ommega2 ] [ post_num ] ), 0.5 * l_val * E_MLIPKJ );
  }
}
__global__ static
void add_grpkj_val ( neuron_t *d_pkj, neuron_solve_t *d_pkj_solve, 
                    int *grpkj_comp, double *grpkj_elem, const int num_grpkj )
{
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  if ( id < num_grpkj )
  {
    int post_num = grpkj_comp [ post_comp * num_grpkj + id ];
	  double l_val = grpkj_elem [ grpkj_val * num_grpkj + id ];
    atomicAdd ( & ( d_pkj_solve -> vec [ cn_gamma2  ] [ post_num ] ), 0.5 * l_val ); // no need atomicAdd ?
    atomicAdd ( & ( d_pkj_solve -> vec [ cn_ommega2 ] [ post_num ] ), 0.5 * l_val * E_GRPKJ );
  }
}

__global__ 
void pkj_cnm_vec_initialize ( neuron_t *d_pkj, neuron_solve_t *d_pkj_solve )
{
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  if ( id < d_pkj -> nc ) 
  {
    double **elem = d_pkj -> elem;
    double **cond = d_pkj -> cond;
    double **ion  = d_pkj -> ion;
    double **vec  = d_pkj_solve -> vec;  
    double l_v_Ca = d_pkj -> rev_ca2 [ id ];
    vec [ cn_gamma1 ] [ id ] = 
                    ( + cond [ g_leak_pkj ] [ id ]
                      + cond [ g_NaF_pkj ] [ id ] * ion [ m_NaF_pkj ] [ id ] * ion [ m_NaF_pkj ] [ id ] * ion [ m_NaF_pkj ] [ id ] * ion [ h_NaF_pkj ] [ id ]
                      + cond [ g_NaP_pkj ] [ id ] * ion [ m_NaP_pkj ] [ id ] * ion [ m_NaP_pkj ] [ id ] * ion [ m_NaP_pkj ] [ id ]
                      + cond [ g_CaP_pkj ] [ id ] * ion [ m_CaP_pkj ] [ id ] * ion [ h_CaP_pkj ] [ id ]
                      + cond [ g_CaT_pkj ] [ id ] * ion [ m_CaT_pkj ] [ id ] * ion [ h_CaT_pkj ] [ id ]
                      + cond [ g_Kh_pkj ] [ id ] * ion [ m_Kh1_pkj ] [ id ]
                      + cond [ g_Kh_pkj ] [ id ] * ion [ m_Kh2_pkj ] [ id ]
                      + cond [ g_Kdr_pkj ] [ id ] * ion [ m_Kdr_pkj ] [ id ] * ion [ m_Kdr_pkj ] [ id ] * ion [ h_Kdr_pkj ] [ id ]
                      + cond [ g_KM_pkj ] [ id ] * ion [ m_KM_pkj ] [ id ]
                      + cond [ g_KA_pkj ] [ id ] * ion [ m_KA_pkj ] [ id ] * ion [ m_KA_pkj ] [ id ] * ion [ m_KA_pkj ] [ id ] * ion [ m_KA_pkj ] [ id ] * ion [ h_KA_pkj ] [ id ]
                      + cond [ g_KC_pkj ] [ id ] * ion [ m_KC_pkj ] [ id ] * ion [ z_KC_pkj ] [ id ] * ion [ z_KC_pkj ] [ id ]
                      + cond [ g_K2_pkj ] [ id ] * ion [ m_K2_pkj ] [ id ] * ion [ z_K2_pkj ] [ id ] * ion [ z_K2_pkj ] [ id ] 
                    ) / 2.0;
    vec [ cn_gamma2 ] [ id ] = 0.0;//= vec [ cn_gamma1 ] [ id ];//
    
    vec [ cn_ommega1 ] [ id ] =
                     ( + cond [ g_leak_pkj ] [ id ] * ( V_LEAK_PKJ ) + elem [ i_ext ] [ id ]
                     + cond [ g_NaF_pkj ] [ id ] * ion [ m_NaF_pkj ] [ id ] * ion [ m_NaF_pkj ] [ id ] * ion [ m_NaF_pkj ] [ id ] * ion [ h_NaF_pkj ] [ id ] * ( V_Na_PKJ )
                     + cond [ g_NaP_pkj ] [ id ] * ion [ m_NaP_pkj ] [ id ] * ion [ m_NaP_pkj ] [ id ] * ion [ m_NaP_pkj ] [ id ] * ( V_Na_PKJ )
                     + cond [ g_CaP_pkj ] [ id ] * ion [ m_CaP_pkj ] [ id ] * ion [ h_CaP_pkj ] [ id ] * ( l_v_Ca )
                     + cond [ g_CaT_pkj ] [ id ] * ion [ m_CaT_pkj ] [ id ] * ion [ h_CaT_pkj ] [ id ] * ( l_v_Ca )
                     + cond [ g_Kh_pkj ] [ id ] * ion [ m_Kh1_pkj ] [ id ] * ( V_KH_PKJ ) //KH????
                     + cond [ g_Kh_pkj ] [ id ] * ion [ m_Kh2_pkj ] [ id ] * ( V_KH_PKJ ) //KH???
                     + cond [ g_Kdr_pkj ] [ id ] * ion [ m_Kdr_pkj ] [ id ] * ion [ m_Kdr_pkj ] [ id ] * ion [ h_Kdr_pkj ] [ id ] * ( V_K_PKJ )
                     + cond [ g_KM_pkj ] [ id ] * ion [ m_KM_pkj ] [ id ] * ( V_K_PKJ )
                     + cond [ g_KA_pkj ] [ id ] * ion [ m_KA_pkj ] [ id ] * ion [ m_KA_pkj ] [ id ] * ion [ m_KA_pkj ] [ id ] * ion [ m_KA_pkj ] [ id ] * ion [ h_KA_pkj ] [ id ] * ( V_K_PKJ )
                     + cond [ g_KC_pkj ] [ id ] * ion [ m_KC_pkj ] [ id ] * ion [ z_KC_pkj ] [ id ] * ion [ z_KC_pkj ] [ id ] * ( V_K_PKJ )
                     + cond [ g_K2_pkj ] [ id ] * ion [ m_K2_pkj ] [ id ] * ion [ z_K2_pkj ] [ id ] * ion [ z_K2_pkj ] [ id ] * ( V_K_PKJ ) 
                    ) / 2.0;
    vec [ cn_ommega2 ] [ id ] = 0.0;//vec [ cn_ommega1 ] [ id ];//
    vec [ cn_v_old ] [ id ] = elem [ v ] [ id ];
  }
  //for ( int i = 0; i < pkj_solve -> nnz; i++ )  { val [ id ] /= 2.0; val_ori [ id ] = val [ id ]; } // to pkj_solve.cu
}
__global__
static void pkj_update_matrix ( neuron_t *d_pkj, neuron_solve_t *d_pkj_solve )
{
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  if ( id < d_pkj -> nc) 
  {
    double **elem = d_pkj -> elem;
    double **cond = d_pkj -> cond;
    double **ion  = d_pkj -> ion;
    double **vec    = d_pkj_solve -> vec;
    double *val     = d_pkj_solve -> val;
    double *val_ori = d_pkj_solve -> val_ori;
    double *b       = d_pkj_solve -> b;
    int    *col  = d_pkj_solve -> col;
    int    *row  = d_pkj_solve -> row;
    double DT = d_pkj -> DT;
    double l_v_Ca = d_pkj -> rev_ca2 [ id ] ;
    vec [ cn_gamma2 ] [ id ] +=  
      ( cond [ g_leak_pkj ] [ id ]
      + cond [ g_NaF_pkj ] [ id ] * ion [ m_NaF_pkj ] [ id ] * ion [ m_NaF_pkj ] [ id ] * ion [ m_NaF_pkj ] [ id ] * ion [ h_NaF_pkj ] [ id ]
      + cond [ g_NaP_pkj ] [ id ] * ion [ m_NaP_pkj ] [ id ] * ion [ m_NaP_pkj ] [ id ] * ion [ m_NaP_pkj ] [ id ]
      + cond [ g_CaP_pkj ] [ id ] * ion [ m_CaP_pkj ] [ id ] * ion [ h_CaP_pkj ] [ id ]
      + cond [ g_CaT_pkj ] [ id ] * ion [ m_CaT_pkj ] [ id ] * ion [ h_CaT_pkj ] [ id ]
      + cond [ g_Kh_pkj ] [ id ] * ion [ m_Kh1_pkj ] [ id ]
      + cond [ g_Kh_pkj ] [ id ] * ion [ m_Kh2_pkj ] [ id ]
      + cond [ g_Kdr_pkj ] [ id ] * ion [ m_Kdr_pkj ] [ id ] * ion [ m_Kdr_pkj ] [ id ] * ion [ h_Kdr_pkj ] [ id ]
      + cond [ g_KM_pkj ] [ id ] * ion [ m_KM_pkj ] [ id ]
      + cond [ g_KA_pkj ] [ id ] * ion [ m_KA_pkj ] [ id ] * ion [ m_KA_pkj ] [ id ] * ion [ m_KA_pkj ] [ id ] * ion [ m_KA_pkj ] [ id ] * ion [ h_KA_pkj ] [ id ]
      + cond [ g_KC_pkj ] [ id ] * ion [ m_KC_pkj ] [ id ] * ion [ z_KC_pkj ] [ id ] * ion [ z_KC_pkj ] [ id ]
      + cond [ g_K2_pkj ] [ id ] * ion [ m_K2_pkj ] [ id ] * ion [ z_K2_pkj ] [ id ] * ion [ z_K2_pkj ] [ id ] 
    ) / 2.0;

    vec [ cn_ommega2 ] [ id ] += 
      ( cond [ g_leak_pkj ] [ id ] * ( V_LEAK_PKJ )  + elem [ i_ext ] [ id ]
      + cond [ g_NaF_pkj ] [ id ] * ion [ m_NaF_pkj ] [ id ] * ion [ m_NaF_pkj ] [ id ] * ion [ m_NaF_pkj ] [ id ] * ion [ h_NaF_pkj ] [ id ] * ( V_Na_PKJ )
      + cond [ g_NaP_pkj ] [ id ] * ion [ m_NaP_pkj ] [ id ] * ion [ m_NaP_pkj ] [ id ] * ion [ m_NaP_pkj ] [ id ] * ( V_Na_PKJ )
      + cond [ g_CaP_pkj ] [ id ] * ion [ m_CaP_pkj ] [ id ] * ion [ h_CaP_pkj ] [ id ] * ( l_v_Ca )
      + cond [ g_CaT_pkj ] [ id ] * ion [ m_CaT_pkj ] [ id ] * ion [ h_CaT_pkj ] [ id ] * ( l_v_Ca )
      + cond [ g_Kh_pkj ] [ id ] * ion [ m_Kh1_pkj ] [ id ] * ( V_KH_PKJ )
      + cond [ g_Kh_pkj ] [ id ] * ion [ m_Kh2_pkj ] [ id ] * ( V_KH_PKJ )
      + cond [ g_Kdr_pkj ] [ id ] * ion [ m_Kdr_pkj ] [ id ] * ion [ m_Kdr_pkj ] [ id ] * ion [ h_Kdr_pkj ] [ id ] * ( V_K_PKJ )
      + cond [ g_KM_pkj ] [ id ] * ion [ m_KM_pkj ] [ id ] * ( V_K_PKJ )
      + cond [ g_KA_pkj ] [ id ] * ion [ m_KA_pkj ] [ id ] * ion [ m_KA_pkj ] [ id ] * ion [ m_KA_pkj ] [ id ] * ion [ m_KA_pkj ] [ id ] * ion [ h_KA_pkj ] [ id ] * ( V_K_PKJ )
      + cond [ g_KC_pkj ] [ id ] * ion [ m_KC_pkj ] [ id ] * ion [ z_KC_pkj ] [ id ] * ion [ z_KC_pkj ] [ id ] * ( V_K_PKJ )
      + cond [ g_K2_pkj ] [ id ] * ion [ m_K2_pkj ] [ id ] * ion [ z_K2_pkj ] [ id ] * ion [ z_K2_pkj ] [ id ] * ( V_K_PKJ ) 
    ) / 2.0;

      int d = d_pkj_solve -> dig [ id ];
      val [ d ] += ( elem [ Cm ] [ id ] / DT) + vec [ cn_gamma2 ] [ id ];
      b [ id ] = 0.0;
      for (int j = row [ id ]; j < row [ id + 1 ]; j++) {
          b [ id ] -= elem [ v ] [ col [ j ] ] * val_ori [ j ];
      }
      b [ id ] += (elem [ Cm ] [ id ] / DT - vec [ cn_gamma1 ] [ id ]) * elem [ v ] [ id ] 
                 + vec [ cn_ommega1 ] [ id ] + vec [ cn_ommega2 ] [ id ];
      vec [ cn_ommega1 ] [ id ] = vec [ cn_ommega2 ] [ id ];
      vec [ cn_gamma1  ] [ id ] = vec [ cn_gamma2  ] [ id ];
      //d_pkj_solve -> vec [ cn_v_old ] [ id ] = elem [ v ] [ id ];
  }
}

__host__
void pkj_solve_by_cnm ( neuron_t *d_pkj, neuron_solve_t *d_pkj_solve, 
                        neuron_t *p_pkj, neuron_solve_t *p_pkj_solve, 
                        synapse_t *d_grpkj, synapse_t *d_mlipkj )
{  
  // global
  int nc = p_pkj -> nc;
  static int numThreadsPerBlock = p_pkj_solve -> numThreadsPerBlock;
  static int numBlocks = p_pkj_solve -> numBlocks;

  // update ion
  pkj_update_ion_2nd <<< numBlocks, numThreadsPerBlock >>> ( d_pkj, d_pkj_solve, CN_DT ); 
  //pkj_update_ion_RK2 <<< numBlocks, numThreadsPerBlock >>> ( d_pkj, d_pkj_solve, CN_DT ); 
  //pkj_update_ion <<< numBlocks, numThreadsPerBlock >>> ( d_pkj, d_pkj_solve, CN_DT ); 

  // reset val and b
  hipMemcpy ( p_pkj_solve -> val,  p_pkj_solve -> val_ori, p_pkj_solve -> nnz * sizeof ( double ), hipMemcpyDeviceToDevice );
  reset_vec <<< numBlocks, numThreadsPerBlock >>> ( d_pkj_solve, nc );
  
  // update val, b and v
  add_grpkj_val <<< ( d_grpkj -> n + 127 ) / 128, 128 >>> 
  ( d_pkj, d_pkj_solve, d_grpkj -> comp, d_grpkj -> elem, d_grpkj -> n );
 // add_mlipkj_val <<< ( d_mlipkj -> n + 127 ) / 128, 128 >>> 
 // ( d_pkj_solve, d_mlipkj -> comp, d_mlipkj -> elem, d_mlipkj -> n );

  pkj_update_matrix <<< numBlocks, numThreadsPerBlock >>> ( d_pkj, d_pkj_solve );
  cg_cusparse_crs ( p_pkj -> nc, p_pkj_solve -> nnz, p_pkj_solve -> val, p_pkj_solve -> col, p_pkj_solve -> row, p_pkj -> elem [ v ], p_pkj_solve -> b );

}

//////////////////////////////// IO /////////////////////////////////

__global__
static void eazy_transposed_matrix ( const int nnz, const double *d_val, double *val_h, int *order )
{
  int i = threadIdx.x + blockDim.x*blockIdx.x;
  if( i < nnz ) { val_h [ i ] = d_val [ order [ i ] ]; }
}

__global__
static void create_transposed_matrix ( const int ngc, const int nnz, 
  const double *d_val, const int *d_col, const int *d_row, 
  double *val_h, int *col_h, int *row_h, int *order )
{
  row_h [ 0 ] = 0;
  int count_row = 0;
  int n_v = 0;
  int n_r = 0;
  for ( int i = 0; i < nnz; i++ ) { order [ i ] = i; }

  for ( int i = 0; i < ngc; i++ ) {
    for ( int j = 0; j < ngc; j++ ) {
      for ( int k = d_row [ j ]; k < d_row [ j + 1 ]; k++ ) {
        if ( d_col [ k ] == i ) {
          val_h [ n_v ] = d_val [ k ];
          order [ n_v ] = k;
          col_h [ n_v ] = j;
          n_v++;          
        }        
      }
    }
    row_h [ n_r + 1 ] = n_v;
    n_r++;
  }
  // Debug
  for ( int i = 0; i < nnz; i++ )
  {
    int j = order [ i ];
    if ( d_val [ j ] != val_h [ i ] )
    {
      printf ("val cpy order error in solve_cnm.cu\n");
    }
  }
  // Debug
  /*
  for ( int i = 0; i < nnz; i++ )
    printf ( "val [ %d ] = %f\n", i, d_val [ i ] );
  for ( int i = 0; i < nnz; i++ ) 
    printf ( "val_h [ %d ] = %f\n", i, val_h [ i ] );
  for ( int i = 0; i < nnz; i++ )
    printf ( "col [ %d ] = %d\n", i, d_col [ i ] );
  for ( int i = 0; i < nnz; i++ ) 
    printf ( "col_h [ %d ] = %d\n", i, col_h [ i ] );
  for ( int i = 0; i < ngc + 1; i++ )
    printf ( "row [ %d ] = %d\n", i, d_row [ i ] );
  for ( int i = 0; i < ngc + 1; i++ )
    printf ( "row_h [ %d ] = %d\n", i, row_h [ i ] );
  */
}

__host__
static void bicg_cusparse_crs( const int ngc, const int nnz, 
  const double *d_val, const int *d_col, const int *d_row, double *d_x, double *d_b)
{
  static double *d_r, *d_p, *d_ap;
  static double *d_rs, *d_ps, *d_atps;
  static double *_buffer;
  static double *val_h;  
  static int *col_h, *row_h, *order;


  static int size = 0;
  if ( size < ngc ) {
    if ( size == 0 ) {
        hipFree ( d_r );
        hipFree ( d_p );
        hipFree ( d_ap );
        hipFree ( d_rs );
        hipFree ( d_ps );
        hipFree ( d_atps );
        hipFree ( _buffer );
        hipFree ( val_h );
        hipFree ( col_h );
        hipFree ( row_h );
        hipFree ( order );
    }
    hipMalloc ( ( double ** ) &d_r,   ngc * sizeof ( double ) );
    hipMalloc ( ( double ** ) &d_p,   ngc * sizeof ( double ) );
    hipMalloc ( ( double ** ) &d_ap,  ngc * sizeof ( double ) );
    hipMalloc ( ( double ** ) &d_rs,  ngc * sizeof ( double ) );
    hipMalloc ( ( double ** ) &d_ps,  ngc * sizeof ( double ) );
    hipMalloc ( ( double ** ) &d_atps,  ngc * sizeof ( double ) );
    hipMalloc ( ( double ** ) &val_h, nnz * sizeof ( double ) );
    hipMalloc ( ( int ** )    &col_h, nnz * sizeof ( int ) );
    hipMalloc ( ( int ** )    &row_h, ( ngc + 1 ) * sizeof ( int ) );
    hipMalloc ( ( int ** )    &order, nnz * sizeof ( int ) );
    hipMalloc ( ( double ** ) &_buffer, ngc * sizeof ( double ) );
    create_transposed_matrix <<< 1, 1 >>>
      ( ngc, nnz, d_val, d_col, d_row, val_h, col_h, row_h, order );

    size = ngc;
  }
  eazy_transposed_matrix  <<< (nnz + 127)/128, 128 >>> ( nnz, d_val, val_h, order );
  //hipDeviceSynchronize();

  //hipMalloc ( ( double ** ) &d_r,  ngc * sizeof ( double ) );
  //hipMalloc ( ( double ** ) &d_p,  ngc * sizeof ( double ) );
  //hipMalloc ( ( double ** ) &d_ap, ngc * sizeof ( double ) );
    
  double bnorm, rnorm_k, rnorm_k1;
  double alpha, beta, pap;
  double epsilon = 1.0e-15;
  double cp1 = 1.0;
  double c0 = 0.0;
  double cm1 = -1.0;
  
  hipblasStatus_t stat1;
  hipblasHandle_t handle1;
  hipsparseStatus_t stat2;
  hipsparseHandle_t handle2;
  hipsparseMatDescr_t descrA;
  stat1 = hipblasCreate(&handle1);
  stat2 = hipsparseCreate(&handle2);
  hipsparseCreateMatDescr(&descrA);
  hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
 
  stat1 = hipblasDscal(handle1, ngc, &c0, d_x, 1);		// x = 0
  stat1 = hipblasDcopy(handle1, ngc, d_b, 1, d_r, 1);	// r = b
  stat1 = hipblasDcopy(handle1, ngc, d_r, 1, d_rs, 1);   // rs = r
  stat1 = hipblasDcopy(handle1, ngc, d_r, 1, d_p, 1);	// p = r
  stat1 = hipblasDcopy(handle1, ngc, d_rs, 1, d_ps, 1);	// ps = rs
  stat1 = hipblasDdot(handle1, ngc, d_b, 1, d_b, 1, &bnorm);	// ||b||
  /**/
  for (int k = 0; k < 100; k++) {
	stat2 = cusparseCsrmvEx(handle2,CUSPARSE_ALG_MERGE_PATH, HIPSPARSE_OPERATION_NON_TRANSPOSE,
	  ngc, ngc, nnz, &cp1, HIP_R_64F, descrA, d_val, HIP_R_64F, d_row, d_col, d_p, HIP_R_64F, 
      &c0, HIP_R_64F, d_ap, HIP_R_64F, HIP_R_64F, _buffer );	    // Ap
    stat2 = cusparseCsrmvEx(handle2,CUSPARSE_ALG_MERGE_PATH, HIPSPARSE_OPERATION_NON_TRANSPOSE,
      ngc, ngc, nnz, &cp1, HIP_R_64F, descrA, val_h, HIP_R_64F, row_h, col_h, d_ps, HIP_R_64F, 
      &c0, HIP_R_64F, d_atps, HIP_R_64F, HIP_R_64F, _buffer );   // Atps
    
    stat1 = hipblasDdot(handle1, ngc, d_r, 1, d_rs, 1, &rnorm_k);		// ( rs, r )
    stat1 = hipblasDdot(handle1, ngc, d_ps, 1, d_ap, 1, &pap);		  	// psAp		
    alpha = rnorm_k / pap;												// alpha		
    stat1 = hipblasDaxpy(handle1, ngc, &alpha, d_p, 1, d_x, 1);			// x += alpha * p
    alpha = -1.0 * alpha;
    stat1 = hipblasDaxpy(handle1, ngc, &alpha, d_ap, 1, d_r, 1);			// r -= alpha * ap
    stat1 = hipblasDaxpy(handle1, ngc, &alpha, d_atps, 1, d_rs, 1);		// rs -= alpha * atps

	stat1 = hipblasDdot(handle1, ngc, d_r, 1, d_rs, 1, &rnorm_k1);		// ( r_k+1, rs_k+1 )
	
	if (sqrt(rnorm_k1) <= epsilon * sqrt(bnorm)) { break; }
	
    // beta
    beta = rnorm_k1 / rnorm_k;
    // p = r + beta * p
    stat1 = hipblasDscal(handle1, ngc, &beta, d_p, 1);
    stat1 = hipblasDaxpy(handle1, ngc, &cp1, d_r, 1, d_p, 1);
    // ps = rs + beta * ps
    stat1 = hipblasDscal(handle1, ngc, &beta, d_ps, 1);
    stat1 = hipblasDaxpy(handle1, ngc, &cp1, d_rs, 1, d_ps, 1);
  }
  hipsparseDestroyMatDescr(descrA);
  hipblasDestroy(handle1);
  hipsparseDestroy(handle2);//
      
  //hipFree ( d_r );
  //hipFree ( d_p );
  //hipFree ( d_ap );
}

__global__ static
void add_io_gap_val ( neuron_solve_t *d_io_solve, int *io_gap_comp, double *io_gap_elem, const int num_io_gap )
{
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  if ( id < num_io_gap ) 
  { 
    double l_val = -1.0 * io_gap_elem [ gap_current * num_io_gap + id ];//-0.5
    int post_num = io_gap_comp [ post_comp_gap  * num_io_gap + id ];    
    atomicAdd ( & ( d_io_solve -> vec [ cn_ommega2 ] [ post_num ] ), ( l_val ) );
  }
}

__global__ 
void io_cnm_vec_initialize ( neuron_t *d_io, neuron_solve_t *d_io_solve )
{
  double **elem = d_io -> elem;
  double **cond = d_io -> cond;
  double **ion  = d_io -> ion;
  double **vec  = d_io_solve -> vec;  
  //double *val         = d_io_solve -> val;
  //double *val_ori     = d_io_solve -> val_ori;
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  if ( id < d_io -> nc) 
  {
    vec [ cn_gamma1 ] [ id ] = 
      (   cond [ g_leak_io ] [ id ]
        + cond [ g_CaL_io  ] [ id ] * ion [ k_CaL_io ] [ id ] * ion [ k_CaL_io ] [ id ] * ion [ k_CaL_io ] [ id ] * ion [ l_CaL_io ] [ id ]
        + cond [ g_Na_io   ] [ id ] * ion [ m_Na_io  ] [ id ] * ion [ m_Na_io  ] [ id ] * ion [ m_Na_io  ] [ id ] * ion [ h_Na_io  ] [ id ]
        + cond [ g_Kdr_io  ] [ id ] * ion [ n_Kdr_io ] [ id ] * ion [ p_Kdr_io ] [ id ] 
        + cond [ g_K_io    ] [ id ] * ion [ x_K_io   ] [ id ] * ion [ x_K_io   ] [ id ] * ion [ x_K_io   ] [ id ] * ion [ x_K_io   ] [ id ] 
        + cond [ g_CaH_io  ] [ id ] * ion [ r_CaH_io ] [ id ] * ion [ r_CaH_io ] [ id ]
        + cond [ g_KCa_io  ] [ id ] * ion [ s_KCa_io ] [ id ]
        + cond [ g_H_io    ] [ id ] * ion [ q_H_io   ] [ id ]  ) / 2.0;//*0.5
    vec [ cn_gamma2 ] [ id ] = 0.0;
    vec [ cn_ommega1 ] [ id ] = 
      (   cond [ g_leak_io ] [ id ] * V_LEAK_IO + elem [ i_ext ] [ id ]
        + cond [ g_CaL_io  ] [ id ] * V_Ca_IO * ion [ k_CaL_io ] [ id ] * ion [ k_CaL_io ] [ id ] * ion [ k_CaL_io ] [ id ] * ion [ l_CaL_io ] [ id ]
        + cond [ g_Na_io   ] [ id ] * V_Na_IO * ion [ m_Na_io  ] [ id ] * ion [ m_Na_io  ] [ id ] * ion [ m_Na_io  ] [ id ] * ion [ h_Na_io  ] [ id ]
        + cond [ g_Kdr_io  ] [ id ] * V_K_IO  * ion [ n_Kdr_io ] [ id ] * ion [ p_Kdr_io ] [ id ] 
        + cond [ g_K_io    ] [ id ] * V_K_IO  * ion [ x_K_io   ] [ id ] * ion [ x_K_io   ] [ id ] * ion [ x_K_io   ] [ id ] * ion [ x_K_io   ] [ id ] 
        + cond [ g_CaH_io  ] [ id ] * V_Ca_IO * ion [ r_CaH_io ] [ id ] * ion [ r_CaH_io ] [ id ]
        + cond [ g_KCa_io  ] [ id ] * V_K_IO  * ion [ s_KCa_io ] [ id ]
        + cond [ g_H_io    ] [ id ] * V_H_IO  * ion [ q_H_io   ] [ id ]   ) / 2.0;//*0.5
    vec [ cn_ommega2 ] [ id ] = 0.0;
    vec [ cn_v_old ] [ id ] = elem [ v ] [ id ];
  }
  //for ( int i = 0; i < io_solve -> nnz; i++ )  { val [ id ] /= 2.0; val_ori [ id ] = val [ id ]; } // to io_solve.cu
}
__global__
static void io_update_matrix ( neuron_t *d_io, neuron_solve_t *d_io_solve )
{
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  if ( id < d_io -> nc) 
  {
    double **elem = d_io -> elem;
    double **cond = d_io -> cond;
    double **ion  = d_io -> ion;
    double **vec    = d_io_solve -> vec;
    double *val     = d_io_solve -> val;
    double *val_ori = d_io_solve -> val_ori;
    double *b       = d_io_solve -> b;
    int    *col  = d_io_solve -> col;
    int    *row  = d_io_solve -> row;
    double DT = d_io -> DT;
    vec [ cn_gamma2 ] [ id ] += 
    (   cond [ g_leak_io ] [ id ]
      + cond [ g_CaL_io  ] [ id ] * ion [ k_CaL_io ] [ id ] * ion [ k_CaL_io ] [ id ] * ion [ k_CaL_io ] [ id ] * ion [ l_CaL_io ] [ id ]
      + cond [ g_Na_io   ] [ id ] * ion [ m_Na_io  ] [ id ] * ion [ m_Na_io  ] [ id ] * ion [ m_Na_io  ] [ id ] * ion [ h_Na_io  ] [ id ]
      + cond [ g_Kdr_io  ] [ id ] * ion [ n_Kdr_io ] [ id ] * ion [ p_Kdr_io ] [ id ] 
      + cond [ g_K_io    ] [ id ] * ion [ x_K_io   ] [ id ] * ion [ x_K_io   ] [ id ] * ion [ x_K_io   ] [ id ] * ion [ x_K_io   ] [ id ] 
      + cond [ g_CaH_io  ] [ id ] * ion [ r_CaH_io ] [ id ] * ion [ r_CaH_io ] [ id ]
      + cond [ g_KCa_io  ] [ id ] * ion [ s_KCa_io ] [ id ]
      + cond [ g_H_io    ] [ id ] * ion [ q_H_io   ] [ id ] ) / 2.0;//*0.5
    vec [ cn_ommega2 ] [ id ] += 
    (   cond [ g_leak_io ] [ id ] * V_LEAK_IO + elem [ i_ext ] [ id ]
      + cond [ g_CaL_io  ] [ id ] * V_Ca_IO * ion [ k_CaL_io ] [ id ] * ion [ k_CaL_io ] [ id ] * ion [ k_CaL_io ] [ id ] * ion [ l_CaL_io ] [ id ]
      + cond [ g_Na_io   ] [ id ] * V_Na_IO * ion [ m_Na_io  ] [ id ] * ion [ m_Na_io  ] [ id ] * ion [ m_Na_io  ] [ id ] * ion [ h_Na_io  ] [ id ]
      + cond [ g_Kdr_io  ] [ id ] * V_K_IO  * ion [ n_Kdr_io ] [ id ] * ion [ p_Kdr_io ] [ id ] 
      + cond [ g_K_io    ] [ id ] * V_K_IO  * ion [ x_K_io   ] [ id ] * ion [ x_K_io   ] [ id ] * ion [ x_K_io   ] [ id ] * ion [ x_K_io   ] [ id ] 
      + cond [ g_CaH_io  ] [ id ] * V_Ca_IO * ion [ r_CaH_io ] [ id ] * ion [ r_CaH_io ] [ id ]
      + cond [ g_KCa_io  ] [ id ] * V_K_IO  * ion [ s_KCa_io ] [ id ]
      + cond [ g_H_io    ] [ id ] * V_H_IO  * ion [ q_H_io   ] [ id ] ) / 2.0;//*0.5

      int d = d_io_solve -> dig [ id ];
      val [ d ] += ( elem [ Cm ] [ id ] / DT) + vec [ cn_gamma2 ] [ id ];
      b [ id ] = 0.0;
      for (int j = row [ id ]; j < row [ id + 1 ]; j++) {
          b [ id ] -= elem [ v ] [ col [ j ] ] * val_ori [ j ];
      }
      b [ id ] += (elem [ Cm ] [ id ] / DT - vec [ cn_gamma1 ] [ id ]) * elem [ v ][ id ] + vec [ cn_ommega1 ] [ id ] + vec [ cn_ommega2 ] [ id ];
      vec [ cn_ommega1 ] [ id ] = vec [ cn_ommega2 ] [ id ];
      vec [ cn_gamma1  ] [ id ] = vec [ cn_gamma2  ] [ id ];
  }
} 
__host__
void io_solve_by_cnm ( neuron_t *d_io, neuron_solve_t *d_io_solve, 
                       neuron_t *p_io, neuron_solve_t *p_io_solve, gap_t* d_io_gap )
{  
  // global
  double **ion  = p_io -> ion;
  double **elem = p_io -> elem;
  int nc = p_io -> nc;
  static int numThreadsPerBlock = p_io_solve -> numThreadsPerBlock;
  static int numBlocks = p_io_solve -> numBlocks;

  // update ion
  io_update_ion_2nd <<< numBlocks, numThreadsPerBlock >>> ( d_io, d_io_solve, CN_DT );
  
  // reset val and b
  hipMemcpy ( p_io_solve -> val,  p_io_solve -> val_ori, p_io_solve -> nnz * sizeof ( double ), hipMemcpyDeviceToDevice );
  reset_vec <<< numBlocks, numThreadsPerBlock >>> ( d_io_solve, nc );

  // update val, b and v
  if ( p_io -> n > 1 )
  {
    io_gap_update <<< ( d_io_gap -> n + 127 ) / 128, 128 >>> 
      ( d_io, d_io_gap -> comp, d_io_gap -> elem, d_io_gap -> n );
    add_io_gap_val <<< ( d_io_gap -> n + 127 ) / 128, 128 >>> 
      ( d_io_solve, d_io_gap -> comp, d_io_gap -> elem, d_io_gap -> n );
  }
  io_update_matrix <<< numBlocks, numThreadsPerBlock >>> ( d_io, d_io_solve );
  bicg_cusparse_crs ( nc, p_io_solve -> nnz, p_io_solve -> val,
    p_io_solve -> col, p_io_solve -> row, p_io -> elem [ v ], p_io_solve -> b );
}
